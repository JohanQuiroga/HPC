#include <hip/hip_runtime.h>
#include <cv.h>
#include <highgui.h>

using namespace cv;

void showImage(Mat &image, const char *window) {
	namedWindow(window, CV_WINDOW_NORMAL);
	imshow(window, image);
}

int main(int argc, char** argv)
{
	char* image_name = argv[1];
	Mat hdr;

	hdr = imread(image_name, CV_LOAD_IMAGE_ANYDEPTH);
	if(argc !=2 || !hdr.data){
	        printf("No image Data \n");
        	return -1;
	}

	if(hdr.empty()) {
		printf("Couldn't find or open the image...\n");
		return -1;
	}

	return 0;
}