#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>

#define BLUE 0
#define GREEN 1
#define RED 2

using namespace cv;

__device__ unsigned char clamp(int pixel) {
	if(pixel < 0) {
		pixel = 0;
	} else if (pixel > 255) {
		pixel = 255;
	}
	return (unsigned char)pixel;
}

__global__ void under_exposed(unsigned char* imageIn, unsigned char* imageOut, int width, int height, int channels) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	
	if((Row < height) && (Col < width)) {
		for(int i=0; i<channels; i++) {
			imageOut[Row*width+Col*channels+i] = clamp(imageIn[Row*width+Col*channels+i] - 51);
			//imageOut[Row*width+Col*channels+i] = 100;
		}
		// imageOut[(Row*width+Col)*3+BLUE] = (int)imageIn[(Row*width+Col)*3+BLUE] - 0.2;
		// imageOut[(Row*width+Col)*3+GREEN] = (int)imageIn[(Row*width+Col)*3+GREEN] - 0.2;
		// imageOut[(Row*width+Col)*3+RED] = (int)imageIn[(Row*width+Col)*3+RED] - 0.2;
	}
}

__global__ void over_exposed(unsigned char* imageIn, unsigned char* imageOut, int width, int height, int channels) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
        int Col = blockIdx.x*blockDim.x + threadIdx.x;

        if((Row < height) && (Col < width)) {
                for(int i=0; i<channels; i++) {
                        imageOut[Row*width+Col*channels+i] = clamp(imageIn[Row*width+Col*channels+i] + 51);
                        //imageOut[Row*width+Col*channels+i] = 100;
                }
                // imageOut[(Row*width+Col)*3+BLUE] = (int)imageIn[(Row*width+Col)*3+BLUE] - 0.2;
                // imageOut[(Row*width+Col)*3+GREEN] = (int)imageIn[(Row*width+Col)*3+GREEN] - 0.2;
                // imageOut[(Row*width+Col)*3+RED] = (int)imageIn[(Row*width+Col)*3+RED] - 0.2;
        }

}

void showImage(Mat &image, const char *window) {
	namedWindow(window, CV_WINDOW_NORMAL);
	imshow(window, image);
}

int main(int argc, char** argv) {
	hipError_t err = hipSuccess;
	char* image_name = argv[1];
	clock_t start_cuda, end_cuda;
	double time_used_cuda;
	int width, height, sizeImage;
	Mat image, image_out_under, image_out_over;
	unsigned char *h_ImageData, *d_ImageData, *d_ImageOut_Under, *d_ImageOut_Over, *h_ImageOut_Under, *h_ImageOut_Over;
	Size imageSize; 

	//printf("Image name: %s\n", image_name);
	image = imread(image_name, 0);
	if(argc !=2 || !image.data){
	        printf("No image Data \n");
        	return -1;
	}

	if(image.empty()) {
		printf("Couldn't find or open the image...\n");
		return -1;
	}

//	printf("Type: %d", image.depth());

	imageSize = image.size();
	width = imageSize.width;
	height = imageSize.height;
	sizeImage = sizeof(unsigned char)*width*height*image.channels();
	// sizeImageGrey = sizeof(unsigned char)*width*height;
	
	h_ImageData = (unsigned char *) malloc (sizeImage);
	h_ImageData = image.data;
	h_ImageOut_Under = (unsigned char *) malloc (sizeImage);
	h_ImageOut_Over = (unsigned char *) malloc (sizeImage);

	err = hipMalloc((void **)&d_ImageData, sizeImage);
	if(err != hipSuccess){
        	printf("Error reservando memoria para d_ImageData\n");
	 	exit(-1);
	}
	err = hipMalloc((void **)&d_ImageOut_Under, sizeImage);
	if(err != hipSuccess){
        	printf("Error reservando memoria para d_ImageOut_Under\n");
	 	exit(-1);
	}

	err = hipMalloc((void **)&d_ImageOut_Over, sizeImage);
	if(err != hipSuccess){
        	printf("Error reservando memoria para d_ImageOut_Over\n");
	 	exit(-1);
	}


	err = hipMemcpy(d_ImageData, h_ImageData, sizeImage, hipMemcpyHostToDevice);
	if(err != hipSuccess){
        	printf("Error copiando los datos de h_ImageData a d_ImageData\n");
	 	exit(-1);
	}
	
	int blockSize = 32;
	dim3 dimBlock(blockSize, blockSize, 1);
	dim3 dimGrid(ceil(width/float(blockSize)), ceil(height/float(blockSize)), 1);
	under_exposed<<<dimGrid, dimBlock>>>(d_ImageData, d_ImageOut_Under, width, height, image.channels());
	hipDeviceSynchronize();
	err = hipMemcpy(h_ImageOut_Under, d_ImageOut_Under, sizeImage, hipMemcpyDeviceToHost);
	if(err != hipSuccess){
        	printf("Error copiando los datos de d_ImageOut_Under a h_ImageOut_Under\n");
	 	exit(-1);
	}

	over_exposed<<<dimGrid, dimBlock>>>(d_ImageData, d_ImageOut_Over, width, height, image.channels());
	hipDeviceSynchronize();
	err = hipMemcpy(h_ImageOut_Over, d_ImageOut_Over, sizeImage, hipMemcpyDeviceToHost);
	if(err != hipSuccess){
        	printf("Error copiando los datos de d_ImageOut_Over a h_ImageOut_Under\n");
	 	exit(-1);
	}

	image_out_under.create(height, width, CV_8UC1);
	image_out_under.data = h_ImageOut_Under;
	imwrite("image_out_under.jpg", image_out_under);

	image_out_over.create(height, width, CV_8UC1);
	image_out_over.data = h_ImageOut_Over;
	imwrite("image_out_over.jpg", image_out_over);

	//printf("Done\n\n");
	//showImage(image, "Image In");
	//showImage(image_out_cuda, "Image out CUDA");
	//showImage(image_out_opencv, "Image out OpenCV");
	//waitKey(0);
	free(h_ImageOut_Under); free(h_ImageOut_Over); hipFree(d_ImageData); hipFree(d_ImageOut_Under); hipFree(d_ImageOut_Over);
	return 0;
}
