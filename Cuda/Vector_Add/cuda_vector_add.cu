#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define TAM 5

__global__ void vecAdd(float* d_A, float* d_B, float* d_C) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<TAM)
		d_C[i] = d_A[i] + d_B[i];
}

void printVector(float* A) {
	for(int i=0; i<TAM; i++) {
		printf("%f ", A[i]);
	}
	printf("\n");
}

int main() {
	hipError_t err;
	int size = TAM*sizeof(float);
	float *h_A = (float *) malloc(size);
	float *h_B = (float *) malloc(size);
	float *h_C = (float *) malloc(size);
	
	float *d_A, *d_B, *d_C;
	
	//init vectors
	for(int i=0; i<TAM; i++) {
		h_A[i]=rand()%10;
		h_B[i]=rand()%10;
		h_C[i]=0;
	}
	
	err = hipMalloc((void **)&d_A, size);
	if(err!=hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_B, size);
	if(err!=hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_C, size);
	if(err!=hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_A, size, hipMemcpyHostToDevice);
	
	vecAdd<<<ceil(TAM/256.0), 256>>>(d_A, d_B, d_C);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	printVector(h_A);
	printVector(h_B);
	printVector(h_C);
	free(h_A); free(h_B); free(h_C);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
	return 0;
}
